#include "hip/hip_runtime.h"
#include <iostream>
#include "mex.h"
#include "omp.h"
#include <cmath>
#include "gpu/mxGPUArray.h"
//#include "splinterp.h"
using namespace std;

#if __CUDA_ARCH__ < 600
template <typename T>
__device__ double atomicAdd(T* address, T val)
{
    unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val +
        __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


template <typename T>
void __global__ updateRec(T*rec_new, long long N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) {
        rec_new[i] = max( 0.0, rec_new[i] );
    }
}

template <typename T>
void __global__ compute_xy_shift( const T*Matrix, const T* shift,  T*x_shift, T*y_shift, int Num_pjs){
    int const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<Num_pjs ) {
        int index = 9*i;
        for (int j=0; j<4; j++){
            x_shift[4*i+j] = Matrix[index+0]*shift[2*j] + Matrix[index+3]*0.0 + Matrix[index+6]*shift[2*j+1] ;
            y_shift[4*i+j] = Matrix[index+1]*shift[2*j] + Matrix[index+4]*0.0 + Matrix[index+7]*shift[2*j+1] ;
        }
    }   
    
}

template <typename T>
void __global__ R1norm(const T *d_vec, double* R1, int N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) {        
        atomicAdd( R1 , (double)abs(d_vec[i]) );
    }
}

template <typename T>
void __global__ R1norm(const T *d_vec, T* R1, int N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) {        
        atomicAdd( R1 , abs(d_vec[i]) );
    }
}

template <typename T>
void __global__ computeCoords(T* coords, const int dimx, const int dimy, const int ncx, const int ncy, int ncz, long long N, long long starting_point){
    long long i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) { 
        i+=starting_point;
        coords[3*i]   =  int(           i%dimx ) - ncx  + 1 ;
        //coords[3*i+1] =  int(  ( i%(dimx*dimy) ) /dimx ) - ncy + 1;
        coords[3*i+1] =  ( int( i/dimx ) ) % dimy - ncy + 1;
        coords[3*i+2] =  int(    i/(dimx*dimy) ) - ncz + 1 ;
    }    
}
//xx_h = mod( ((0:692)'),7)-3;
//yy_h = floor( mod( (0:692)', 7*9) /7)-4; yy_h = mod( floor( (0:692)' /7), 9 )-4;
//zz_h = floor( (0:692)'/(7*9)) - 5;

template <typename T>
void __global__ setValue(T*residual, double val, int N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    //T o_ratio_inv = 1.0/o_ratio;
    if (i<N) {
        residual[i] = val;
    }
}

static const int blockSize = 1024;
static const int gridSize = 24; 
template <typename T>
__global__ void sumCommMultiBlock(const T *gArr, long long arraySize, T *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    T sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += abs(gArr[i]);
    __shared__ T shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

template <typename T>
__host__ double sumArray(const T* arr, long long wholeArraySize) {
    T* arr_pt;
    hipMalloc((void**)&arr_pt, wholeArraySize * sizeof(T));
    hipMemcpy(arr_pt, arr, wholeArraySize * sizeof(T), hipMemcpyHostToDevice);

    T sum;
    T* sum_pt;
    hipMalloc((void**)&sum_pt, sizeof(T)*gridSize);
    
    sumCommMultiBlock<<<gridSize, blockSize>>>(arr_pt, wholeArraySize, sum_pt);
    //dev_out now holds the partial result
    sumCommMultiBlock<<<1, blockSize>>>(sum_pt, gridSize, sum_pt);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();
    
    hipMemcpy(&sum, sum_pt, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(arr_pt);
    hipFree(sum_pt);
    return sum;
}




template <typename T>
void __global__ computeResidual( 
          T* residual,
    const T* d_projection,
    const long long N )
{
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N) {
        residual[i] = residual[i]  - d_projection[i];
    }
}

template <typename T>
void __global__ RadonTF(const T*data,const T*Matrix, const int nrows, const int ncols,const T*nc, 
const int o_ratio,const T*x_s,const T*y_s, T* result, const T alpha, long long N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    //long long nrow_cols = nrows*ncols;
    int origin_offset = 1;
    long s ;  
    //#pragma omp parallel for default(shared) private(i,s) schedule(static)
    if (i<N){
        const T & data_i = data[i];
        const T coord_x = int(           i%nrows ) - nc[0]  + 1 ;
        const T coord_y = ( int( i/nrows ) ) % ncols - nc[1] + 1;
        const T coord_z =  int(    i/(nrows*ncols) ) - nc[2] + 1 ;

        //long index = i*3;
        const T x_i = Matrix[0]*coord_x + Matrix[3]*coord_y + Matrix[6]*coord_z + nc[0];
        const T y_i = Matrix[1]*coord_x + Matrix[4]*coord_y + Matrix[7]*coord_z + nc[1];

        for (s=0; s<o_ratio; s++){
            //for (i = 0; i < N; ++i) {
            T x_is = x_i + x_s[s] - origin_offset;
            T y_is = y_i + y_s[s] - origin_offset;

            // get coordinates of bounding grid locations
            long long x_1 = ( long long) floor(x_is) ;
            long long x_2 = x_1 + 1;
            long long y_1 = ( long long) floor(y_is) ;
            long long y_2 = y_1 + 1;
            
            if (x_1>=-1 && x_2<=nrows  &&  y_1>=-1 && y_2<=ncols ){ 
                T w_x1 = x_2 - x_is ;
                T w_x2 = 1   - w_x1;
                T w_y1 = y_2 - y_is ;
                T w_y2 = 1   - w_y1;            
                if (x_1==-1){
                    if (y_1==-1){
                        atomicAdd( &result[x_2 + y_2*nrows] , alpha * w_x2*w_y2 * data_i );
                    }
                    else if(y_2==ncols){
                        atomicAdd( &result[x_2 + y_1*nrows] , alpha * w_x2*w_y1 * data_i );
                    }
                    else{
                        atomicAdd( &result[x_2 + y_1*nrows] , alpha * w_x2*w_y1 * data_i );
                        atomicAdd( &result[x_2 + y_2*nrows] , alpha * w_x2*w_y2 * data_i );                    
                    }
                }
                else if (x_2==nrows){
                    if (y_1==-1){
                        atomicAdd( &result[x_1 + y_2*nrows] , alpha * w_x1*w_y2 * data_i );
                    }
                    else if(y_2==ncols){
                        atomicAdd( &result[x_1 + y_1*nrows] , alpha * w_x1*w_y1 * data_i );
                    }
                    else{
                        atomicAdd( &result[x_1 + y_1*nrows] , alpha * w_x1*w_y1 * data_i );
                        atomicAdd( &result[x_1 + y_2*nrows] , alpha * w_x1*w_y2 * data_i );                  
                    } 
                }
                else{
                    if (y_1==-1){
                        atomicAdd( &result[x_1 + y_2*nrows] , alpha * w_x1*w_y2 * data_i );
                        atomicAdd( &result[x_2 + y_2*nrows] , alpha * w_x2*w_y2 * data_i );
                    }
                    else if(y_2==ncols){
                        atomicAdd( &result[x_1 + y_1*nrows] , alpha * w_x1*w_y1 * data_i );
                        atomicAdd( &result[x_2 + y_1*nrows] , alpha * w_x2*w_y1 * data_i );
                    }
                    else{
                        atomicAdd( &result[x_1 + y_1*nrows] , alpha * w_x1*w_y1 * data_i );
                        atomicAdd( &result[x_1 + y_2*nrows] , alpha * w_x1*w_y2 * data_i );
                        atomicAdd( &result[x_2 + y_1*nrows] , alpha * w_x2*w_y1 * data_i );
                        atomicAdd( &result[x_2 + y_2*nrows] , alpha * w_x2*w_y2 * data_i );                  
                    }                               
                }
            }
        }
    }
}


template <typename T>
void __global__ RadonTpose_updateRec(const T* Matrix, const int nrows, const int ncols, const T* nc, const T* data, 
const int o_ratio, const T*x_s,const T*y_s, T* Rec, const bool* support, float dt, long long N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    int origin_offset = 1;
    long s;
    //#pragma omp parallel for default(shared) private(s) schedule(static)  
    if( i < N  && support[i]) {
        const T coord_x = int(           i%nrows ) - nc[0]  + 1 ;
        const T coord_y = ( int( i/nrows ) ) % ncols - nc[1] + 1;
        const T coord_z =  int(    i/(nrows*ncols) ) - nc[2] + 1 ;

        //long index = i*3;
        const T x0 = Matrix[0]*coord_x + Matrix[3]*coord_y + Matrix[6]*coord_z + nc[0];
        const T y0 = Matrix[1]*coord_x + Matrix[4]*coord_y + Matrix[7]*coord_z + nc[1];
        for (s=0; s<o_ratio; s++){
            T x_i = x0 + x_s[s];
            T y_i = y0 + y_s[s];
            // get coordinates of bounding grid locations
            long long x_1 = ( long long) floor(x_i) - origin_offset;
            long long x_2 = x_1 + 1;
            long long y_1 = ( long long) floor(y_i) - origin_offset;
            long long y_2 = y_1 + 1;
            
            // handle special case where x/y is the last element
            if ( (x_i - origin_offset) == (nrows-1) )   { x_2 -= 1; x_1 -= 1;}
            if ( (y_i - origin_offset) == (ncols-1) )   { y_2 -= 1; y_1 -= 1;}
            
            // return 0 for target values that are out of bounds
            if (x_1 < 0 | x_2 > (nrows - 1) |  y_1 < 0 | y_2 > (ncols - 1)){
                //result[i] = 0;
            }
            else {
                // get the array values
                const T& f_11 = data[x_1 + y_1*nrows];
                const T& f_12 = data[x_1 + y_2*nrows];
                const T& f_21 = data[x_2 + y_1*nrows];
                const T& f_22 = data[x_2 + y_2*nrows];
                
                // compute weights
                T w_x1 = x_2 - (x_i - origin_offset);
                T w_x2 = (x_i - origin_offset) - x_1;
                T w_y1 = y_2 - (y_i - origin_offset);
                T w_y2 = (y_i - origin_offset) - y_1;
                
                T a,b;
                a = f_11 * w_x1 + f_21 * w_x2;
                b = f_12 * w_x1 + f_22 * w_x2;
                Rec[i] -= dt*(a * w_y1 + b * w_y2);
            }
        }
    }
}


void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[]) {
    int const threadsPerBlock = 256;
    int blocksPerGridRec;  //blocksPerGridPrj

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();    
    hipError_t err = hipSuccess;
    err = hipSetDevice(0);            // Set device 0 as current    
    if(err!=hipSuccess){
        printf("cuda fail to set\n");
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
        device, deviceProp.major, deviceProp.minor);
    }

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";    
    if ((nrhs!=5)&&(nrhs!=6) ){ //|| (nlhs!=1)  ) {
        cout << "number of parameter not correct"<<endl;
        mexErrMsgIdAndTxt(errId, errMsg); //!(mxIsGPUArray(prhs[0]))
    }
    /*
    * 0: projections
    * 1: mat
    * 2: coord
    * 3: nc
    * 4: rec
    * 5: x_shift3
    * 6: y_shift3
    * 7: z_shift3
    */

    float const * rec1          = mxGetSingles(prhs[0]);
    float const * rec2          = mxGetSingles(prhs[1]);
    float const * rec3          = mxGetSingles(prhs[2]);
    float const * Matrix        = mxGetSingles(prhs[3]);
    float const * alpha         = mxGetSingles(prhs[4]);
  

    const size_t o_ratio=4;
    const mwSize* recSize   = (mxGetDimensions(prhs[0]));
    const mwSize dimx    = recSize[0];
    const mwSize dimy    = recSize[1];
    const mwSize dimz    = recSize[2];

    const mwSize* dims_Mat   = (mxGetDimensions(prhs[3]));
    const mwSize Num_pjs     = max(mwSize (1),dims_Mat[2]); 
    const mwSize* dims_alpha = (mxGetDimensions(prhs[4]));
    const mwSize Num_pjs2    = max(mwSize (1),dims_alpha[1]); 

    const mwSize dims_pjs[] = {dimx,dimy, Num_pjs}; 
    
    const long long nrow_cols  = dimx*dimy;
    const long long nPjsPoints = dimx*dimy*Num_pjs;
    const long long recPoints  = dimx*dimy*dimz;
    //cout << dimx <<", " << dimy <<", " << dimz <<", "  << endl;

    if ( Num_pjs!=Num_pjs2 || dims_Mat[0]!=3 || dims_Mat[1]!=3 || dims_alpha[0]!=3 ){
        cout << "dimension not matched."<<endl;
        mexErrMsgIdAndTxt(errId, errMsg); //!(mxIsGPUArray(prhs[0]))
    }

    if(mxGetClassID(prhs[0]) == mxDOUBLE_CLASS || mxGetClassID(prhs[1]) == mxDOUBLE_CLASS
    || mxGetClassID(prhs[2]) == mxDOUBLE_CLASS  ){     
        printf("can't work with double\n");
        return;
    }    

    const mwSize ncx = int (floor(dimx/2.0)+1);
    const mwSize ncy = int (floor(dimy/2.0)+1);
    const mwSize ncz = int (floor(dimz/2.0)+1);
    

    // copy rotation matrix to GPU
    float * d_Matrix;
    hipMalloc( &d_Matrix, 9*Num_pjs*sizeof(float) );
    hipMemcpy(  d_Matrix, Matrix, 9*Num_pjs*sizeof(float), hipMemcpyHostToDevice );
    

    // create reconstruciton on GPU
    float * d_rec1, *d_rec2, *d_rec3;
    hipMalloc( &d_rec1, recPoints*sizeof(float) );
    hipMalloc( &d_rec2, recPoints*sizeof(float) );
    hipMalloc( &d_rec3, recPoints*sizeof(float) );
    hipMemcpy( d_rec1, rec1, recPoints*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_rec2, rec2, recPoints*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( d_rec3, rec3, recPoints*sizeof(float), hipMemcpyHostToDevice);

    
    // create residual on GPU
    float* d_residual;
    hipMalloc( &d_residual, nPjsPoints*sizeof(float) ) ;
    hipMemset(  d_residual, 0, nPjsPoints*sizeof(float) );        
    //hipMemset( d_residual, 0, nPjsPoints*sizeof(float) );

    //blocksPerGrid    = (recPoints + threadsPerBlock - 1) / threadsPerBlock;
    //blocksPerGridPrj = (nPjsPoints + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGridRec = (recPoints + threadsPerBlock - 1) / threadsPerBlock;


    // compute rotated shift
    float shift[]  = {0.25,0.25, 0.25,-0.25,-0.25,0.25,-0.25,-0.25};
    float *shift_ptr;    
    hipMalloc( (void**) &shift_ptr, 8*sizeof(float) );
    hipMemcpy( shift_ptr, shift, 8*sizeof(float), hipMemcpyHostToDevice);
    float * x_shift, *y_shift;
    hipMalloc( (void**) &x_shift, 4*Num_pjs*sizeof(float) );
    hipMalloc( (void**) &y_shift, 4*Num_pjs*sizeof(float) );
    compute_xy_shift<<<2, threadsPerBlock>>>( d_Matrix, shift_ptr, x_shift, y_shift, Num_pjs );
    float const *d_xs2         = (float  *) x_shift;
    float const *d_ys2         = (float  *) y_shift;

    // compute cartesian coordinates
    //computeCoords<<<blocksPerGridRec,threadsPerBlock>>>(d_Coord, dimx, dimy, ncx,ncy, ncz, recPoints, 0);

    // compute nc = [ncx,ncy,ncz]
    //const float nc_cpu[]  = { ncx,ncy,ncz}; 
    const float nc_cpu[]  = { float(floor(dimx/2.0)+1), float(floor(dimy/2.0)+1), float(floor(dimz/2.0)+1)};     
    float * d_nc;
    hipMalloc( (void**)&d_nc, 3*sizeof(float) );
    hipMemcpy( d_nc, nc_cpu, 3*sizeof(float), hipMemcpyHostToDevice ); 


    // iteration


        // compute forward projection        
    for (int i = 0; i < Num_pjs; i++){
            RadonTF<<<blocksPerGridRec, threadsPerBlock>>>(d_rec1, d_Matrix + i*9, dimx,dimy,d_nc,
            o_ratio,d_xs2+i*o_ratio,d_ys2+i*o_ratio, d_residual + i*nrow_cols, alpha[3*i]/o_ratio, recPoints);
            
            RadonTF<<<blocksPerGridRec, threadsPerBlock>>>(d_rec2, d_Matrix + i*9, dimx,dimy,d_nc,
            o_ratio,d_xs2+i*o_ratio,d_ys2+i*o_ratio, d_residual + i*nrow_cols, alpha[3*i+1]/o_ratio, recPoints);
            
            RadonTF<<<blocksPerGridRec, threadsPerBlock>>>(d_rec3, d_Matrix + i*9, dimx,dimy,d_nc,
            o_ratio,d_xs2+i*o_ratio,d_ys2+i*o_ratio, d_residual + i*nrow_cols, alpha[3*i+2]/o_ratio, recPoints);
    }

    plhs[0] = mxCreateNumericArray(3, dims_pjs, mxSINGLE_CLASS, mxREAL);
    float* residual   = mxGetSingles(plhs[0]);   
    hipMemcpy( residual, d_residual, nPjsPoints*sizeof(float), hipMemcpyDeviceToHost);     
           

    /* return result  */
    hipFree(d_rec1);
    hipFree(d_rec2);
    hipFree(d_rec3);
    hipFree(d_residual);
    hipFree(d_Matrix);
    hipFree(d_nc);
    hipFree( shift_ptr );
    hipFree( x_shift );
    hipFree( y_shift );
    
    
}


/*
// compute diffnorm of xyshift
float x_norm, x_norm2,  *x_pt, *x_pt2;
hipMalloc( (void**)&x_pt, sizeof(float) );
hipMalloc( (void**)&x_pt2, sizeof(float) );
hipMemset(x_pt, 0, sizeof(float));
hipMemset(x_pt2, 0, sizeof(float));
R1norm<<<2, threadsPerBlock>>>(d_xs,  x_pt, Num_pjs*4);
R1norm<<<2, threadsPerBlock>>>(d_xs2,  x_pt2, Num_pjs*4);
hipMemcpy( &x_norm ,  x_pt, sizeof(float), hipMemcpyDeviceToHost ) ;
hipMemcpy( &x_norm2 ,  x_pt2, sizeof(float), hipMemcpyDeviceToHost ) ;
hipFree( x_pt);
cout << "x_norm = " << x_norm << ", x_norm2 = " << x_norm2 <<endl;
*/



















