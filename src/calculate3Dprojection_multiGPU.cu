#include "hip/hip_runtime.h"
#include <iostream>
#include "mex.h"
#include "omp.h"
#include <cmath>
#include "gpu/mxGPUArray.h"
//#include "splinterp.h"
using namespace std;

#if __CUDA_ARCH__ < 600
template <typename T>
__device__ double atomicAdd(T* address, T val)
{
    unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val +
        __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


template <typename T>
void __global__ updateRec(T*rec_new, long long N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) {
        rec_new[i] = max( 0.0, rec_new[i] );
    }
}

template <typename T>
void __host__ compute_xy_shift( const T*Matrix, const T* shift,  T*x_shift, T*y_shift, int Num_pjs){
    for (int i=0; i<Num_pjs; i++ ) {
        int index = 9*i;
        for (int j=0; j<4; j++){
            x_shift[4*i+j] = Matrix[index+0]*shift[2*j] + Matrix[index+3]*0.0 + Matrix[index+6]*shift[2*j+1] ;
            y_shift[4*i+j] = Matrix[index+1]*shift[2*j] + Matrix[index+4]*0.0 + Matrix[index+7]*shift[2*j+1] ;
        }
    }   
}

template <typename T>
void __global__ R1norm(const T *d_vec, double* R1, int N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) {        
        atomicAdd( R1 , (double)abs(d_vec[i]) );
    }
}

template <typename T>
void __global__ R1norm(const T *d_vec, T* R1, int N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) {        
        atomicAdd( R1 , abs(d_vec[i]) );
    }
}
/*
static const int blockSize = 1024;
static const int gridSize = 24; 
template <typename T>
__global__ void sumCommMultiBlock(const T *gArr, long long arraySize, T *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    T sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += abs(gArr[i]);
    __shared__ T shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

template <typename T>
__host__ double sumArray(const T* arr, long long wholeArraySize) {
    T* arr_pt;
    hipMalloc((void**)&arr_pt, wholeArraySize * sizeof(T));
    hipMemcpy(arr_pt, arr, wholeArraySize * sizeof(T), hipMemcpyHostToDevice);

    T sum;
    T* sum_pt;
    hipMalloc((void**)&sum_pt, sizeof(T)*gridSize);
    
    sumCommMultiBlock<<<gridSize, blockSize>>>(arr_pt, wholeArraySize, sum_pt);
    //dev_out now holds the partial result
    sumCommMultiBlock<<<1, blockSize>>>(sum_pt, gridSize, sum_pt);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();
    
    hipMemcpy(&sum, sum_pt, sizeof(T), hipMemcpyDeviceToHost);
    hipFree(arr_pt);
    hipFree(sum_pt);
    return sum;
}
*/

template <typename T>
void __global__ computeCoords(T* coords, const int dimx, const int dimy, const int ncx, const int ncy, int ncz, long long N, long long starting_point){
    long long i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N ) { 
        i+=starting_point;
        coords[3*i]   =  int(           i%dimx ) - ncx  + 1 ;
        //coords[3*i+1] =  int(  ( i%(dimx*dimy) ) /dimx ) - ncy + 1;
        coords[3*i+1] =  ( int( i/dimx ) ) % dimy - ncy + 1;
        coords[3*i+2] =  int(    i/(dimx*dimy) ) - ncz + 1 ;
    }    
}
//xx_h = mod( ((0:692)'),7)-3;
//yy_h = floor( mod( (0:692)', 7*9) /7)-4; yy_h = mod( floor( (0:692)' /7), 9 )-4;
//zz_h = floor( (0:692)'/(7*9)) - 5;

template <typename T>
void __global__ setValue(T*residual, double val, int N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    //T o_ratio_inv = 1.0/o_ratio;
    if (i<N) {
        residual[i] = val;
    }
}

template <typename T>
void __global__ residualAdd (T*residual_0, T*residual_i, long long N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N) {
        residual_0[i] += residual_i[i];
    }    
}

template <typename T>
void __global__ computeResidual(T*residual, const T scale, long long N){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N) {
        residual[i] = residual[i]*scale;
    }
}

template <typename T>
void __global__ RadonTF(const T*data,const T*Matrix, const int dimx, const int dimy,const T*nc, const T*nc_proj, 
const int o_ratio,const T*x_s,const T*y_s, T*result, const int dimx_proj, const int dimy_proj, long long N, long long starting_pt){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    //long long nrow_cols = dimx*dimy;
    int origin_offset = 1;
    long s ;  
    //#pragma omp parallel for default(shared) private(i,s) schedule(static)
    if (i<N){
        const T & data_i = data[i];
        long long ii = i+starting_pt;        
        const T coord_x = int(           ii%dimx ) - nc[0]  + 1 ;
        const T coord_y = ( int( ii/dimx ) ) % dimy - nc[1] + 1;
        const T coord_z =  int(    ii/(dimx*dimy) ) - nc[2] + 1 ;

        //long index = i*3;
        const T x_i = Matrix[0]*coord_x + Matrix[3]*coord_y + Matrix[6]*coord_z + nc_proj[0];
        const T y_i = Matrix[1]*coord_x + Matrix[4]*coord_y + Matrix[7]*coord_z + nc_proj[1];

        for (s=0; s<o_ratio; s++){
            //for (i = 0; i < N; ++i) {
            T x_is = x_i + x_s[s] - origin_offset;
            T y_is = y_i + y_s[s] - origin_offset;

            // get coordinates of bounding grid locations
            long long x_1 = ( long long) floor(x_is) ;
            long long x_2 = x_1 + 1;
            long long y_1 = ( long long) floor(y_is) ;
            long long y_2 = y_1 + 1;
            
            if (x_1>=-1 && x_2<=dimx_proj  &&  y_1>=-1 && y_2<=dimy_proj ){ 
                T w_x1 = x_2 - x_is ;
                T w_x2 = 1   - w_x1;
                T w_y1 = y_2 - y_is ;
                T w_y2 = 1   - w_y1;            
                if (x_1==-1){
                    if (y_1==-1){
                        atomicAdd( &result[x_2 + y_2*dimx_proj] , w_x2*w_y2 * data_i);
                    }
                    else if(y_2==dimy_proj){
                        atomicAdd( &result[x_2 + y_1*dimx_proj] , w_x2*w_y1 * data_i);
                    }
                    else{
                        atomicAdd( &result[x_2 + y_1*dimx_proj] , w_x2*w_y1 * data_i);
                        atomicAdd( &result[x_2 + y_2*dimx_proj] , w_x2*w_y2 * data_i);                    
                    }
                }
                else if (x_2==dimx_proj){
                    if (y_1==-1){
                        atomicAdd( &result[x_1 + y_2*dimx_proj] , w_x1*w_y2 * data_i);
                    }
                    else if(y_2==dimy_proj){
                        atomicAdd( &result[x_1 + y_1*dimx_proj] , w_x1*w_y1 * data_i);
                    }
                    else{
                        atomicAdd( &result[x_1 + y_1*dimx_proj] , w_x1*w_y1 * data_i);
                        atomicAdd( &result[x_1 + y_2*dimx_proj] , w_x1*w_y2 * data_i);                  
                    } 
                }
                else{
                    if (y_1==-1){
                        atomicAdd( &result[x_1 + y_2*dimx_proj] , w_x1*w_y2 * data_i);
                        atomicAdd( &result[x_2 + y_2*dimx_proj] , w_x2*w_y2 * data_i);
                    }
                    else if(y_2==dimy_proj){
                        atomicAdd( &result[x_1 + y_1*dimx_proj] , w_x1*w_y1 * data_i);
                        atomicAdd( &result[x_2 + y_1*dimx_proj] , w_x2*w_y1 * data_i);
                    }
                    else{
                        atomicAdd( &result[x_1 + y_1*dimx_proj] , w_x1*w_y1 * data_i);
                        atomicAdd( &result[x_1 + y_2*dimx_proj] , w_x1*w_y2 * data_i);
                        atomicAdd( &result[x_2 + y_1*dimx_proj] , w_x2*w_y1 * data_i);
                        atomicAdd( &result[x_2 + y_2*dimx_proj] , w_x2*w_y2 * data_i);                  
                    }                               
                }
            }
        }
    }
}


template <typename T>
void __global__ RadonTpose_updateRec(const T* Matrix, const int dimx, const int dimy, const T* nc, const T* data, 
const int o_ratio, const T*x_s,const T*y_s, T* Rec, float dt, long long N, long long starting_pt){
    long long const i = blockDim.x * blockIdx.x + threadIdx.x;
    int origin_offset = 1;
    long s;
    //#pragma omp parallel for default(shared) private(s) schedule(static)  
    if( i < N ) {
        long long ii = i+starting_pt;
        const T coord_x = int(           ii%dimx )  - nc[0] + 1 ;
        const T coord_y = ( int( ii/dimx ) ) % dimy - nc[1] + 1;
        const T coord_z =  int(    ii/(dimx*dimy) ) - nc[2] + 1 ;

        //long index = i*3;
        const T x0 = Matrix[0]*coord_x + Matrix[3]*coord_y + Matrix[6]*coord_z + nc[0];
        const T y0 = Matrix[1]*coord_x + Matrix[4]*coord_y + Matrix[7]*coord_z + nc[1];
        for (s=0; s<o_ratio; s++){
            T x_i = x0 + x_s[s];
            T y_i = y0 + y_s[s];
            // get coordinates of bounding grid locations
            long long x_1 = ( long long) floor(x_i) - origin_offset;
            long long x_2 = x_1 + 1;
            long long y_1 = ( long long) floor(y_i) - origin_offset;
            long long y_2 = y_1 + 1;
            
            // handle special case where x/y is the last element
            if ( (x_i - origin_offset) == (dimx-1) )   { x_2 -= 1; x_1 -= 1;}
            if ( (y_i - origin_offset) == (dimy-1) )   { y_2 -= 1; y_1 -= 1;}
            
            // return 0 for target values that are out of bounds
            if (x_1 < 0 | x_2 > (dimx - 1) |  y_1 < 0 | y_2 > (dimy - 1)){
                //result[i] = 0;
            }
            else {
                // get the array values
                const T& f_11 = data[x_1 + y_1*dimx];
                const T& f_12 = data[x_1 + y_2*dimx];
                const T& f_21 = data[x_2 + y_1*dimx];
                const T& f_22 = data[x_2 + y_2*dimx];
                
                // compute weights
                T w_x1 = x_2 - (x_i - origin_offset);
                T w_x2 = (x_i - origin_offset) - x_1;
                T w_y1 = y_2 - (y_i - origin_offset);
                T w_y2 = (y_i - origin_offset) - y_1;
                
                T a,b;
                a = f_11 * w_x1 + f_21 * w_x2;
                b = f_12 * w_x1 + f_22 * w_x2;
                Rec[i] -= dt*(a * w_y1 + b * w_y2);
            }
        }
    }
}

void mexFunction(int nlhs, mxArray *plhs[],int nrhs, const mxArray *prhs[]) {
    int const threadsPerBlock = 256;
    int blocksPerGridPrj, blocksPerGridRec_i;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();    
    hipError_t err = hipSuccess;
    err = hipSetDevice(0);            // Set device 0 as current    
    if(err!=hipSuccess){
        printf("cuda fail to set\n");
    }

    int deviceCount; //int const deviceCount=2;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        //printf("Device %d has compute capability %d.%d.\n",
        //device, deviceProp.major, deviceProp.minor);
    }

    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";    
    if (nrhs!=2 && nrhs!=3){ //|| (nlhs!=1)  ) {
        cout << "number of parameter not recognized"<<endl;
        mexErrMsgIdAndTxt(errId, errMsg); //!(mxIsGPUArray(prhs[0]))
    }
    /*
    * 0: projections
    * 1: mat
    * 2: coord
    * 3: nc
    * 4: rec
    * 5: x_shift3
    * 6: y_shift3
    * 7: z_shift3
    */

    float const * rec     = mxGetSingles(prhs[0]);
    float const * Matrix  = mxGetSingles(prhs[1]);    

    const size_t o_ratio=4;
    const mwSize* rec_size   = (mxGetDimensions(prhs[0]));
    const mwSize dimx    = rec_size[0];
    const mwSize dimy    = rec_size[1];
    const mwSize dimz    = rec_size[2];
    const mwSize dimz_i  = dimz/deviceCount;    
    //dimz = dimz_i*deviceCount;

    const mwSize* dims_Mat = (mxGetDimensions(prhs[1]));
    const mwSize Num_pjs   = max(mwSize (1),dims_Mat[2]);   //should be 1
    //cout << "Number of projection = " <<Num_pjs <<endl;
    const mwSize R1 = dims_Mat[0];
    const mwSize R2 = dims_Mat[1];
    
    mwSize dimx_proj, dimy_proj;
    if(nrhs==3){
        const double * dims_proj_pt = mxGetDoubles(prhs[2]);
        dimx_proj = mwSize( dims_proj_pt[0] );
        dimy_proj = mwSize( dims_proj_pt[1] );        
    }  
    else{
        dimx_proj = dimx;
        dimy_proj = dimy;
    }
    
    const long long nrow_cols   = dimx_proj*dimy_proj;
    const long long nPjsPoints  = dimx_proj*dimy_proj*Num_pjs;
    const long long recPoints_i = dimz_i*dimx*dimy;
    const mwSize proj_size[] = {dimx_proj,dimy_proj,Num_pjs};
    //const mwSize recSize_i[] = {dimx,dimy,dimz_i};

    const mwSize ncx = int (floor(dimx/2.0)+1);
    const mwSize ncy = int (floor(dimy/2.0)+1);
    const mwSize ncz = int (floor(dimz/2.0)+1);

    //mexPrintf("%d\n",npoints);  

    blocksPerGridPrj     = (nPjsPoints  + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGridRec_i   = (recPoints_i + threadsPerBlock - 1) / threadsPerBlock;

    if( R1!=3 || R2!=3 ){
        cout << "dimension not matched"<<endl;
        
    }
    if(mxGetClassID(prhs[0]) == mxDOUBLE_CLASS || mxGetClassID(prhs[1]) == mxDOUBLE_CLASS ){     
        printf("can't work with double\n");
        mexErrMsgIdAndTxt(errId, errMsg);
    }   
    

    const float nc_proj[] = { float(floor(dimx_proj/2.0)+1), float(floor(dimy_proj/2.0)+1)};
    
    hipDeviceEnablePeerAccess(1,0);
    hipDeviceEnablePeerAccess(0,0); 

    //int z_thickness = dimz/deviceCount;
    //long long z_pts = z_thickness*dimx*dimy;

    // compute rotated shift
    float shift[]  = {0.25,0.25, 0.25,-0.25,-0.25,0.25,-0.25,-0.25};
    float * x_shift = new float[4*Num_pjs], *y_shift = new float[4*Num_pjs];
    compute_xy_shift( Matrix, shift, x_shift, y_shift, Num_pjs );

    // copy rotation matrix to GPU
    float ** d_Matrix   = new float*[deviceCount];
    float ** d_residual = new float*[deviceCount];
    float  * d_residual_temp;
    float ** d_x_shift  = new float*[deviceCount];
    float ** d_y_shift  = new float*[deviceCount];
    const float nc[]    = { float(ncx), float(ncy), float(ncz)}; 
    float ** d_nc       = new float*[deviceCount];
    float ** d_nc_proj  = new float*[deviceCount];    
    float ** d_rec      = new float*[deviceCount]; 

    hipMalloc( &d_residual_temp,    nPjsPoints*sizeof(float) );
    hipMemset(  d_residual_temp, 0, nPjsPoints*sizeof(float) );

    
    // copy data to the 1st GPU: projections, Matrix, residual
    /*
    float * d_projections;
    hipMalloc( &d_projections,    nPjsPoints*sizeof(float) );
    hipMemset(  d_projections, 0, nPjsPoints*sizeof(float) );
    //hipMemcpy(  d_projections, projections, nPjsPoints*sizeof(float), hipMemcpyHostToDevice );
    */

    // plhs[0] is the returning reconstruction
    plhs[0] = mxCreateNumericArray(3, proj_size, mxSINGLE_CLASS, mxREAL);
    float*    residual   = (float*)mxGetSingles(plhs[0]);        


    for(int device=0; device<deviceCount; device++) {
        hipSetDevice(device); 
        
        hipMalloc( &d_Matrix[device], 9*Num_pjs*sizeof(float) );
        hipMemcpy(  d_Matrix[device], Matrix, 9*Num_pjs*sizeof(float), hipMemcpyHostToDevice );   

        hipMalloc( &d_residual[device],    nPjsPoints*sizeof(float) );
        hipMemset(  d_residual[device], 0, nPjsPoints*sizeof(float) );

        hipMalloc( &d_x_shift[device], 4*Num_pjs*sizeof(float) );
        hipMemcpy(  d_x_shift[device], x_shift, 4*Num_pjs*sizeof(float), hipMemcpyHostToDevice ); 

        hipMalloc( &d_y_shift[device], 4*Num_pjs*sizeof(float) );
        hipMemcpy(  d_y_shift[device], y_shift, 4*Num_pjs*sizeof(float), hipMemcpyHostToDevice ); 

        hipMalloc( &d_nc[device], 3*sizeof(float) );
        hipMemcpy(  d_nc[device], nc, 3*sizeof(float), hipMemcpyHostToDevice ); 
        
        hipMalloc( &d_nc_proj[device], 2*sizeof(float) );
        hipMemcpy(  d_nc_proj[device], nc_proj, 2*sizeof(float), hipMemcpyHostToDevice );
        
        hipMalloc( &d_rec[device],    recPoints_i*sizeof(float) );
        hipMemset(  d_rec[device], 0, recPoints_i*sizeof(float) );
    }


    for(int device=0; device<deviceCount; device++){
        hipSetDevice(device); 
        hipMemcpy( d_rec[device], rec + recPoints_i*device, recPoints_i*sizeof(float), hipMemcpyHostToDevice);
    }             

    
    // compute calculated projections from rec1 & rec2   
    hipSetDevice(0);
    hipMemset( d_residual[0], 0, nPjsPoints*sizeof(float) );
    for(int device=1; device<deviceCount; device++){
        hipSetDevice(device);       
        hipMemset( d_residual[device], 0, nPjsPoints*sizeof(float) );
        for (int i = 0; i < Num_pjs; i++){
            RadonTF<<<blocksPerGridRec_i, threadsPerBlock>>>(d_rec[device], d_Matrix[device] + i*9, dimx,dimy,d_nc[device],d_nc_proj[device],
            o_ratio,d_x_shift[device]+i*o_ratio,d_y_shift[device]+i*o_ratio, d_residual[device] + i*nrow_cols, dimx_proj,dimy_proj, recPoints_i, recPoints_i*device);
        }
        hipMemcpyPeer( d_residual_temp, 0, d_residual[device], device,  nPjsPoints*sizeof(float));
        hipSetDevice(0);
        residualAdd<<<blocksPerGridPrj, threadsPerBlock>>>(d_residual[0], d_residual_temp, nPjsPoints);
    }

    hipSetDevice(0);                
    for (int i = 0; i < Num_pjs; i++){
        RadonTF<<<blocksPerGridRec_i, threadsPerBlock>>>(d_rec[0], d_Matrix[0] + i*9, dimx,dimy,d_nc[0],d_nc_proj[0],
        o_ratio,d_x_shift[0]+i*o_ratio,d_y_shift[0]+i*o_ratio, d_residual[0] + i*nrow_cols, dimx_proj,dimy_proj, recPoints_i,0);
    }        
    
    // compute residual: = forward projection - measure projections
    computeResidual<<<blocksPerGridPrj, threadsPerBlock>>>(d_residual[0],1.0f/o_ratio, nPjsPoints);
    hipMemcpy( residual, d_residual[0],  nPjsPoints*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_residual_temp);
    for (int i=0; i<deviceCount; i++){
        hipFree(d_rec[i]);
        hipFree(d_residual[i]);
        hipFree(d_Matrix[i]);
        //hipFree(d_projections);
        hipFree(d_nc[i]);
        hipFree(d_x_shift[i]);
        hipFree(d_y_shift[i]);
    }
    
}



















